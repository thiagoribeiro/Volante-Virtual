#include "hip/hip_runtime.h"
/* Copyright (c) 2007, University of North Carolina at Chapel Hill
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the <organization> nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY University of North Carolina at Chapel Hill ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL University of North Carolina at Chapel Hill BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <cutil.h>

#define COMPUTE_BOTH_DEPTHMAPS
#define DOWNLOAD_BOTH_DEPTHMAPS
#define PERFORM_CONSISTENCY_CHECK
#define CONSISTENCY_THRESHOLD  1
/*ORIGINAL
#define IMAGE_ALIGN                 64

#define STEREO_MIND                 0
#define STEREO_MAXD                 55
#define STEREO_WINSIZE_W            7
#define STEREO_WINSIZE_H            7

#define STEREO_SHARED_W             96
#define STEREO_SHARED_H             38
#define STEREO_RADIUS_W             (STEREO_WINSIZE_W/2)
#define STEREO_RADIUS_H             (STEREO_WINSIZE_H/2)
#define STEREO_DRANGE               (STEREO_MAXD-STEREO_MIND+1)
#define STEREO_APRON_W              64
#define STEREO_APRON_H              (2*STEREO_RADIUS_H)
#define STEREO_BLOCK_W              (STEREO_SHARED_W-STEREO_APRON_W)
#define STEREO_BLOCK_H              (STEREO_SHARED_H-STEREO_APRON_H)
#define STEREO_THREADS_W            8
#define STEREO_THREADS_H            STEREO_SHARED_H
#define STEREO_SHARED_MEM           (STEREO_SHARED_W*STEREO_SHARED_H*2+STEREO_THREADS_W*STEREO_THREADS_H*4)
*/
/*TESTE1
#define IMAGE_ALIGN                 64

#define STEREO_MIND                 0
#define STEREO_MAXD                 127
#define STEREO_WINSIZE_W            7
#define STEREO_WINSIZE_H            7

#define STEREO_SHARED_W             96
#define STEREO_SHARED_H             38
#define STEREO_RADIUS_W             (STEREO_WINSIZE_W/2)
#define STEREO_RADIUS_H             (STEREO_WINSIZE_H/2)
#define STEREO_DRANGE               (STEREO_MAXD-STEREO_MIND+1)
#define STEREO_APRON_W              64
#define STEREO_APRON_H              (2*STEREO_RADIUS_H)
#define STEREO_BLOCK_W              (STEREO_SHARED_W-STEREO_APRON_W)
#define STEREO_BLOCK_H              (STEREO_SHARED_H-STEREO_APRON_H)
#define STEREO_THREADS_W            8
#define STEREO_THREADS_H            STEREO_SHARED_H
#define STEREO_SHARED_MEM           (STEREO_SHARED_W*STEREO_SHARED_H*2+STEREO_THREADS_W*STEREO_THREADS_H*4)
*/
/*TESTE2
#define IMAGE_ALIGN                 64

#define STEREO_MIND                 0
#define STEREO_MAXD                 127
#define STEREO_WINSIZE_W            11
#define STEREO_WINSIZE_H            11

#define STEREO_SHARED_W             96
#define STEREO_SHARED_H             38
#define STEREO_RADIUS_W             (STEREO_WINSIZE_W/2)
#define STEREO_RADIUS_H             (STEREO_WINSIZE_H/2)
#define STEREO_DRANGE               (STEREO_MAXD-STEREO_MIND+1)
#define STEREO_APRON_W              64
#define STEREO_APRON_H              (2*STEREO_RADIUS_H)
#define STEREO_BLOCK_W              (STEREO_SHARED_W-STEREO_APRON_W)
#define STEREO_BLOCK_H              (STEREO_SHARED_H-STEREO_APRON_H)
#define STEREO_THREADS_W            8
#define STEREO_THREADS_H            STEREO_SHARED_H
#define STEREO_SHARED_MEM           (STEREO_SHARED_W*STEREO_SHARED_H*2+STEREO_THREADS_W*STEREO_THREADS_H*4)
*/
#define IMAGE_ALIGN                 10

#define STEREO_MIND                 0
#define STEREO_MAXD                 55
#define STEREO_WINSIZE_W            7
#define STEREO_WINSIZE_H            7

#define STEREO_SHARED_W             96
#define STEREO_SHARED_H             38
#define STEREO_RADIUS_W             (STEREO_WINSIZE_W/2)
#define STEREO_RADIUS_H             (STEREO_WINSIZE_H/2)
#define STEREO_DRANGE               (STEREO_MAXD-STEREO_MIND+1)
#define STEREO_APRON_W              64
#define STEREO_APRON_H              (2*STEREO_RADIUS_H)
#define STEREO_BLOCK_W              (STEREO_SHARED_W-STEREO_APRON_W)
#define STEREO_BLOCK_H              (STEREO_SHARED_H-STEREO_APRON_H)
#define STEREO_THREADS_W            8
#define STEREO_THREADS_H            STEREO_SHARED_H
#define STEREO_SHARED_MEM           (STEREO_SHARED_W*STEREO_SHARED_H*2+STEREO_THREADS_W*STEREO_THREADS_H*4)

static int g_w;
static int g_h;
static int g_alignW;
static unsigned char *g_imageLeft;
static unsigned char *g_imageRight;
static float *g_disparityLeft;
static float *g_disparityRight;


__device__ int diff( int l, int r )
{
    return abs(l-r);
    //return (l-r)*(l-r);
}

__device__ float subpixel( int c0, int c1, int c2 )
{
    float denom,doff;
    denom = 2*(c0-2*c1+c2);
    if(denom<1e-2 || c1>c0 || c1>c2) {
        return 0;
    } else {
        doff = (c0-c2)/denom;
        return doff;
    }
}

__global__ void stereo( float *disparityLeft,
                        float *disparityRight,
                        const unsigned char *left,
                        const unsigned char *right,
                        size_t width )
{
    extern __shared__ unsigned char sdata[];
    unsigned char *sleft = sdata;
    unsigned char *sright = sdata + STEREO_SHARED_W*STEREO_SHARED_H;
    unsigned int *stemp = (unsigned int*)(sdata + STEREO_SHARED_W*STEREO_SHARED_H*2);
    float bestd[4];
    unsigned int sum;
    int ii,it,is,i;
    unsigned int lastcs[3];
    unsigned int bestcs[3];
    int d;

    // Read image blocks into shared memory.
    const int si = __mul24(threadIdx.y,STEREO_SHARED_W) + 4*threadIdx.x;
    const int gi = __mul24(__mul24(blockIdx.y,STEREO_BLOCK_H) + threadIdx.y,width) + __mul24(blockIdx.x,STEREO_BLOCK_W) + 4*threadIdx.x;
    *(unsigned int*)(sleft+si)                     = *(unsigned int*)(left+gi);
    *(unsigned int*)(sleft+si+4*STEREO_THREADS_W)  = *(unsigned int*)(left+gi+4*STEREO_THREADS_W);
    *(unsigned int*)(sleft+si+8*STEREO_THREADS_W)  = *(unsigned int*)(left+gi+8*STEREO_THREADS_W);
    *(unsigned int*)(sright+si)                     = *(unsigned int*)(right+gi);
    *(unsigned int*)(sright+si+4*STEREO_THREADS_W)  = *(unsigned int*)(right+gi+4*STEREO_THREADS_W);
    *(unsigned int*)(sright+si+8*STEREO_THREADS_W)  = *(unsigned int*)(right+gi+8*STEREO_THREADS_W);
    __syncthreads();

    // Do left/right matching with separable box filter.
    for(int pix=0; pix<4; pix++) {
        ii = __mul24(threadIdx.y,STEREO_SHARED_W)+STEREO_APRON_W-STEREO_RADIUS_W+4*threadIdx.x+pix;
        it = __mul24(threadIdx.y,STEREO_THREADS_W)+threadIdx.x;
        bestcs[0] = bestcs[1] = bestcs[2] = INT_MAX;
        for(d=STEREO_MIND; d<=STEREO_MAXD; d++) {
            sum = 0;
            // Horizontal sum.
            for(is=ii-STEREO_RADIUS_W; is<=ii+STEREO_RADIUS_W; is++) {
                sum += diff(sleft[is],sright[is-d]);
            }
            stemp[it] = sum;
            __syncthreads();
            if(threadIdx.y>=STEREO_RADIUS_H && threadIdx.y<STEREO_SHARED_H-STEREO_RADIUS_H) {
                // Vertical sum.
                sum = 0;
                is = it-STEREO_RADIUS_H*STEREO_THREADS_W;
                for(i=-STEREO_RADIUS_H; i<=STEREO_RADIUS_H; i++,is+=STEREO_THREADS_W) {
                    sum += stemp[is];
                }
                // Best.
                if(d==STEREO_MIND) {
                    lastcs[1] = lastcs[2] = sum;
                } else {
                    lastcs[0] = lastcs[1];
                    lastcs[1] = lastcs[2];
                    lastcs[2] = sum;
                    if(lastcs[1] < bestcs[1]) {
                        bestcs[0] = lastcs[0];
                        bestcs[1] = lastcs[1];
                        bestcs[2] = lastcs[2];
                        bestd[pix] = d-1;
                    }
                }
            }
        }
        bestd[pix] += subpixel(bestcs[0],bestcs[1],bestcs[2]);
    }
    // Write results.
    if(threadIdx.y>=STEREO_RADIUS_H && threadIdx.y<STEREO_SHARED_H-STEREO_RADIUS_H) {
		ii = blockIdx.x*STEREO_BLOCK_W + STEREO_APRON_W - STEREO_RADIUS_W + 4*threadIdx.x;
		it = blockIdx.y*STEREO_BLOCK_H + threadIdx.y;
		is = __mul24(it,width) + ii;
		*(float*)(disparityLeft+is+0) = bestd[0];
		*(float*)(disparityLeft+is+1) = bestd[1];
		*(float*)(disparityLeft+is+2) = bestd[2];
		*(float*)(disparityLeft+is+3) = bestd[3];
    }

#ifdef COMPUTE_BOTH_DEPTHMAPS
    // Do right/left matching with separable box filter.
    for(int pix=0; pix<4; pix++) {
        ii = __mul24(threadIdx.y,STEREO_SHARED_W)+STEREO_RADIUS_W+4*threadIdx.x+pix;
        it = __mul24(threadIdx.y,STEREO_THREADS_W)+threadIdx.x;
        bestcs[0] = bestcs[1] = bestcs[2] = INT_MAX;
        for(d=STEREO_MIND; d<=STEREO_MAXD; d++) {
            sum = 0;
            // Horizontal sum.
            for(is=ii-STEREO_RADIUS_W; is<=ii+STEREO_RADIUS_W; is++) {
                sum += diff(sleft[is+d],sright[is]);
            }
            stemp[it] = sum;
            __syncthreads();
            if(threadIdx.y>=STEREO_RADIUS_H && threadIdx.y<STEREO_SHARED_H-STEREO_RADIUS_H) {
                // Vertical sum.
                sum = 0;
                is = it-STEREO_RADIUS_H*STEREO_THREADS_W;
                for(i=-STEREO_RADIUS_H; i<=STEREO_RADIUS_H; i++,is+=STEREO_THREADS_W) {
                    sum += stemp[is];
                }
                // Best.
                if(d==STEREO_MIND) {
                    lastcs[1] = lastcs[2] = sum;
                } else {
                    lastcs[0] = lastcs[1];
                    lastcs[1] = lastcs[2];
                    lastcs[2] = sum;
                    if(lastcs[1] < bestcs[1]) {
                        bestcs[0] = lastcs[0];
                        bestcs[1] = lastcs[1];
                        bestcs[2] = lastcs[2];
                        bestd[pix] = d-1;
                    }
                }
            }
        }
        bestd[pix] += subpixel(bestcs[0],bestcs[1],bestcs[2]);
    }
    // Write results.
    if(threadIdx.y>=STEREO_RADIUS_H && threadIdx.y<STEREO_SHARED_H-STEREO_RADIUS_H) {
		ii = blockIdx.x*STEREO_BLOCK_W + STEREO_RADIUS_W + 4*threadIdx.x;
		it = blockIdx.y*STEREO_BLOCK_H + threadIdx.y;
		is = __mul24(it,width) + ii;
		*(float*)(disparityRight+is+0) = bestd[0];
		*(float*)(disparityRight+is+1) = bestd[1];
		*(float*)(disparityRight+is+2) = bestd[2];
		*(float*)(disparityRight+is+3) = bestd[3];
    }
#endif //COMPUTE_BOTH_DEPTHMAPS
}

__global__ void consistency( float *left,
							 float *right,
							 size_t width,
							 int consist )
{
    const int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    const int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
    const int i = __mul24(y,width)+x;
	int dl = (int)left[i];
	int dlr = (int)right[i-dl];
#ifdef DOWNLOAD_BOTH_DEPTHMAPS
	int dr = (int)right[i];
	int drl = (int)left[i+dr];
#endif
	__syncthreads();
	if(abs(dl-dlr)>consist)
		left[i] = 255;
#ifdef DOWNLOAD_BOTH_DEPTHMAPS
	if(abs(dr-drl)>consist)
		right[i] = 255;
#endif
}

int align( int n, int a )
{
    int r = n % a;
    if(r==0)
        return n;
    else
        return n-r+a;
}

extern "C" void stereoInit( int w, int h )
{
    size_t pitch;
    hipChannelFormatDesc fmt = hipCreateChannelDesc<unsigned int>();

    g_w = w;
    g_h = h;
    g_alignW = align(w,IMAGE_ALIGN);

    CUDA_SAFE_CALL(hipMallocPitch((void**)&g_imageLeft,&pitch,g_alignW,h));
    CUDA_SAFE_CALL(hipMallocPitch((void**)&g_imageRight,&pitch,g_alignW,h));
    CUDA_SAFE_CALL(hipMallocPitch((void**)&g_disparityLeft,&pitch,g_alignW*sizeof(float),h));
    CUDA_SAFE_CALL(hipMallocPitch((void**)&g_disparityRight,&pitch,g_alignW*sizeof(float),h));
	CUDA_SAFE_CALL(hipMemset(g_disparityLeft,0,g_alignW*sizeof(float)*h));
	CUDA_SAFE_CALL(hipMemset(g_disparityRight,0,g_alignW*sizeof(float)*h));
}

extern "C" void stereoUpload( const unsigned char *left, const unsigned char *right )
{
    CUDA_SAFE_CALL(hipMemcpy2D(g_imageLeft,g_alignW,left,g_w,g_w,g_h,
        hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy2D(g_imageRight,g_alignW,right,g_w,g_w,g_h,
        hipMemcpyHostToDevice));
}

int divUp( int n, int d )
{
    if(n%d==0)
        return n/d;
    else
        return n/d + 1;
}

extern "C" void stereoProcess()
{
    // Disparity map computation.
    dim3 threads(STEREO_THREADS_W,STEREO_THREADS_H);
    dim3 grid((g_alignW-STEREO_APRON_W)/STEREO_BLOCK_W,g_h/STEREO_BLOCK_H);
    stereo<<<grid,threads,STEREO_SHARED_MEM>>>(g_disparityLeft,g_disparityRight,
        g_imageLeft,g_imageRight,g_alignW);

#if defined(PERFORM_CONSISTENCY_CHECK) && defined(COMPUTE_BOTH_DEPTHMAPS)
    // Consistency check.
    dim3 cthreads(16,16);
    dim3 cgrid(g_alignW/cthreads.x,g_h/cthreads.y);
    consistency<<<cgrid,cthreads>>>(g_disparityLeft,g_disparityRight,g_alignW,
        CONSISTENCY_THRESHOLD);
#endif
}

extern "C" void stereoDownload( float *disparityLeft, float *disparityRight )
{
    CUDA_SAFE_CALL(hipMemcpy2D(disparityLeft,g_w*sizeof(float),g_disparityLeft,g_alignW*sizeof(float),g_w*sizeof(float),g_h,
        hipMemcpyDeviceToHost));
#if defined(DOWNLOAD_BOTH_DEPTHMAPS) && defined(COMPUTE_BOTH_DEPTHMAPS)
    CUDA_SAFE_CALL(hipMemcpy2D(disparityRight,g_w*sizeof(float),g_disparityRight,g_alignW*sizeof(float),g_w*sizeof(float),g_h,
        hipMemcpyDeviceToHost));
#endif
}
